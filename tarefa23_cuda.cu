
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>

/*
Ana Laura Fernandes de Oliveira
Larissa Domingues Gomes
Pedro Henrique Lima Carvalho
Pedro Henrique Reis Rodrigues
Tárcila Fernanda Resende da Silva


Tarefa 23

Tempo do Programa Serial
real  1m50.604s
user  1m15.349s
sys   0m0.108s

Tempo do Programa Paralelo CPU
real  0m40.297s
user  1m11.810s
sys   0m0.315s

Tempo do Programa Paralelo GPU - OpenMP
real  0m1.284s
user  0m0.143s
sys   0m0.814s

Tempo do Programa Paralelo GPU - Cuda
real  0m0.957s
user  0m0.132s
sys   0m0.742s

*** Obs: Rodamos novamente o Paralelo GPU - OpenMP para obter novo tempo hoje
*** Parcode hoje está mais rápido.


Speedup Paralelo GPU-OpenMp em relação Paralelo CPU: 2,44

Speedup Paralelo GPU-Cuda em relação Paralelo CPU: 42,107 


------------ Métricas OpenMp ----------------
Métricas da Nvdia
Invocations                                Event Name         Min         Max         Avg       Total
Device "GeForce GT 1030 (0)"
    Kernel: mm$_omp_fn$0
          1                            warps_launched      155832      155832      155832      155832

==11722== Metric result:
Invocations                               Metric Name                        Metric Description         Min         Max         Avg
Device "GeForce GT 1030 (0)"
    Kernel: mm$_omp_fn$0
          1                 warp_execution_efficiency                 Warp Execution Efficiency      86.81%      86.81%      86.81%


------------- Métricas Cuda ----------------

*** Em função de problemas com parcode não foi possível obter as métricas

*/

__global__ void mm_cuda(double* a, double* b, double* c, int width){
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   int row = blockIdx.y * blockDim.y + threadIdx.y;

   if(col < width && row < width){
      double sum = 0;
      
      for (int i=0; i<width; i++){
         double x = a[row * width + i];
         double y = b[i*width + col];
         sum += x*y; 
      }
      c[row * width + col] = sum;
   }
} 

int main()
{
  int width = 2000;
  double *a = (double*) malloc (width * width * sizeof(double));
  double *b = (double*) malloc (width * width * sizeof(double));
  double *c = (double*) malloc (width * width * sizeof(double));

  for(int i = 0; i < width; i++) {
    for(int j = 0; j < width; j++) {
      a[i*width+j] = i;
      b[i*width+j] = j;
      c[i*width+j] = 0;
    }
  }

  int size = width*width*sizeof(double);
  double *d_a, *d_b, *d_c;
 
  hipMalloc((void **) &d_a, size);
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_b, size);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  hipMalloc((void **) &d_c, size);
  
  int blockSize = 2000;
  dim3 dimGrid ((width-1)/blockSize+1, (width-1)/blockSize+1, 1);
  dim3 dimBlock (blockSize, blockSize, 1);

  mm_cuda<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

//    for(int i = 0; i < width; i++) {
//    for(int j = 0; j < width; j++) {
//      printf("\n c[%d][%d] = %f",i,j,c[i*width+j]);
//    }
//   }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
